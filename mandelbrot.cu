
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <err.h>
#include <stdint.h>

#include <X11/Xlib.h>
#include <X11/Xutil.h>
#include <omp.h>

#include <stdlib.h>
#include <X11/keysym.h>
#include <fcntl.h>

#define CPU_CORES 8
#define BENCHMARK_ITERATIONS 50
int Version = 0;

static int dim = 800;
static int n = 512;
static int m = 512;
static int max_iter = 100;
static uint32_t *colors;
uint32_t *device_colors;

double total_time;
int instances;

#ifdef BENCHMARK
double single_core;
#endif
// X11 data 
#ifdef SHOW_X
static Display *dpy;
static XImage *bitmap;
static Window win;
static Atom wmDeleteMessage;
static GC gc;

//destroy window and x variables 
static void exit_x11(void){
#ifdef BENCHMARK
    printf("\nThe Average time for this version was %f over %d instances.\n", total_time/instances, instances);
#endif
	XDestroyWindow(dpy, win);
	XCloseDisplay(dpy);
}


// create Xwindow 
static void init_x11(){
	// Attempt to open the display 
	dpy = XOpenDisplay(NULL);
	
	// Failure
	if (!dpy) exit(0);
	
	uint32_t long white = WhitePixel(dpy,DefaultScreen(dpy));
	uint32_t long black = BlackPixel(dpy,DefaultScreen(dpy));
	

	win = XCreateSimpleWindow(dpy, DefaultRootWindow(dpy),
            0, 0, dim, dim, 0, black, white);
	
	// We want to be notified when the window appears 
	XSelectInput(dpy, win, StructureNotifyMask);
	
	// Make it appear 
	XMapWindow(dpy, win);
	
	while (1){
        XEvent e;
		XNextEvent(dpy, &e);
		if (e.type == MapNotify) break;
	}
	
	XTextProperty tp;
    char name1[128] = "Mandelbrot Single Core";
    char* n = name1;
	Status st = XStringListToTextProperty(&n, 1, &tp);
	if (st) XSetWMName(dpy, win, &tp);

	// Wait for the MapNotify event 
	XFlush(dpy);
    int depth = DefaultDepth(dpy, DefaultScreen(dpy));    
    Visual *visual = DefaultVisual(dpy, DefaultScreen(dpy));

    bitmap = XCreateImage(dpy, visual, depth, ZPixmap, 0,
            (char*) malloc(dim * dim * 32), dim, dim, 32, 0);

	// Init GC 
	gc = XCreateGC(dpy, win, 0, NULL);
	XSetForeground(dpy, gc, black);
	
	XSelectInput(dpy, win, ExposureMask | ButtonPressMask | Button3MotionMask | KeyPressMask | StructureNotifyMask);
	
	wmDeleteMessage = XInternAtom(dpy, "WM_DELETE_WINDOW", False);
	XSetWMProtocols(dpy, win, &wmDeleteMessage, 1);
}
#endif

//create colors used to draw the mandelbrot set 
void init_colours(void) {
    float freq = 6.3 / max_iter;
	for (int i = 0; i < max_iter; i++){
        char r = sin(freq * i + 3) * 127 + 128;
        char g = sin(freq * i + 5) * 127 + 128;
        char b = sin(freq * i + 1) * 127 + 128;
		
		colors[i] = b + 256 * g + 256 * 256 * r;
	}
	
	colors[max_iter] = 0;
}

void checkErr(hipError_t err, char* msg){
    if (err != hipSuccess){
        fprintf(stderr, "%s (error code %d: '%s'", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}



// Here we have the single core version for mandel_double
uint32_t mandel_double_single(double cr, double ci, int max_iter) {
    double zr = 0, zi = 0, zrsqr = 0, zisqr = 0;

    uint32_t i;

    for (i = 0; i < max_iter; i++){
		zi = zr * zi;
		zi += zi;
		zi += ci;
		zr = zrsqr - zisqr + cr;
		zrsqr = zr * zr;
		zisqr = zi * zi;
		
    //the fewer iterations it takes to diverge, the farther from the set
		if (zrsqr + zisqr > 4.0) break;
    }
    return i;
}

void mandel_single(uint32_t *counts, double xmin, double ymin,
            double step, int max_iter, int dim, uint32_t *colors) {
    int i, x, y;
    double cr, ci;
    if (Version == 1){
        # pragma omp parallel for num_threads(CPU_CORES) \
            schedule(runtime) private(i,x,y,cr,ci) \
            shared(counts, dim, colors, xmin, ymin, step, max_iter)
        for (i = 1; i < dim*dim; i++){
            x = i % dim;
            y = i / dim;
            cr = xmin + x * step;
            ci = ymin + y * step;
            counts[y * dim + x]  = colors[mandel_double_single(cr, ci, max_iter)];
        }
    } else {
        for (i = 1; i < dim*dim; i++){
            x = i % dim;
            y = i / dim;
            cr = xmin + x * step;
            ci = ymin + y * step;
            counts[y * dim + x]  = colors[mandel_double_single(cr, ci, max_iter)];
        }
    }
}

double display_double_single(double xcen, double ycen, double scale,
        uint32_t *host_counts, uint32_t *colors){
    dim3 numBlocks(dim,dim);
    double xmin = xcen - (scale/2);
    double ymin = ycen - (scale/2);
    double step = scale / dim;

#ifdef BENCHMARK
    double start = omp_get_wtime();
#endif 
    mandel_single(host_counts, xmin, ymin, step, max_iter, dim, colors);

#ifdef SHOW_X
    memcpy(bitmap->data, host_counts, dim * dim * sizeof(uint32_t));
#else
    void *data = malloc(dim * dim * sizeof(uint32_t));
    memcpy(data, host_counts, dim * dim * sizeof(uint32_t));
#endif

#ifdef BENCHMARK
    double stop = omp_get_wtime();
    //if (Version == 0) printf("Version: Single Core\tTime: %f\r", stop - start);
    //else printf("Version: OpenMP\t\tTime: %f\r", stop - start);
    fflush(stdout);
#endif

#ifdef SHOW_X
    XPutImage(dpy, win, gc, bitmap,
        0, 0, 0, 0,
        dim, dim);
    XFlush(dpy); 
#endif
#ifdef BENCHMARK
    return stop-start;
#endif
    return 0;
}




/* the mandelbrot set is defined as all complex numbers c such that the 
   equation z = z^2 + c remains bounded. In practice, we calculate max_iter
   iterations of this formula and if the magnitude of z is < 2 we assume it
   is in the set. The greater max_iters the more accurate our representation */
__device__ uint32_t mandel_double(double cr, double ci, int max_iter) {
    double zr = 0;
    double zi = 0;
    double zrsqr = 0;
    double zisqr = 0;

    uint32_t i;

    for (i = 0; i < max_iter; i++){
		zi = zr * zi;
		zi += zi + ci;
		zr = zrsqr - zisqr + cr;
		zrsqr = zr * zr;
		zisqr = zi * zi;
		
    //the fewer iterations it takes to diverge, the farther from the set
		if (zrsqr + zisqr > 4.0) break;
    }
	
    return i;
}

/* turn each x y coordinate into a complex number and run the mandelbrot formula on it */
__global__ void mandel_kernel(uint32_t *counts, double xmin, double ymin,
            double step, int max_iter, int dim, uint32_t *colors) {
    int pix_per_thread = dim * dim / (gridDim.x * blockDim.x);
    int tId = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = pix_per_thread * tId;
    for (int i = offset; i < offset + pix_per_thread; i++){
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = colors[mandel_double(cr, ci, max_iter)];
    }
    if (gridDim.x * blockDim.x * pix_per_thread < dim * dim
            && tId < (dim * dim) - (blockDim.x * gridDim.x)){
        int i = blockDim.x * gridDim.x * pix_per_thread + tId;
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x]  = colors[mandel_double(cr, ci, max_iter)];
    }
}

/* For each point, evaluate its colour */
double display_double_cuda(double xcen, double ycen, double scale,
        uint32_t *dev_counts, uint32_t *colors){
    dim3 numBlocks(dim,dim);
    
    double xmin = xcen - (scale/2);
    double ymin = ycen - (scale/2);
    double step = scale / dim;
    hipError_t err = hipSuccess;

#ifdef BENCHMARK
    double start = omp_get_wtime();
#endif 

    mandel_kernel<<<n, m>>>(dev_counts, xmin , ymin, step, max_iter, dim, colors);
    checkErr(err, (char*)"Failed to run Kernel");
#ifdef SHOW_X
    err = cudaMemcpy(bitmap->data, dev_counts, dim * dim * sizeof(uint32_t), cudaMemcpyDeviceToHost);
#else
    void *data = malloc(dim * dim * sizeof(uint32_t));
    err = hipMemcpy(data, dev_counts, dim * dim * sizeof(uint32_t), hipMemcpyDeviceToHost);
#endif
    checkErr(err, (char*)"Failed to copy dev_counts back");

#ifdef BENCHMARK
    double stop = omp_get_wtime();
    //printf("Version: CUDA\t\tTime: %f\r", stop - start);
    fflush(stdout);
#endif
#ifdef SHOW_X
    XPutImage(dpy, win, gc, bitmap,
        0, 0, 0, 0,
        dim, dim);
    XFlush(dpy); 
#endif
#ifdef BENCHMARK
    return stop-start;
#endif
    return 0;
}

void display_double(double xcen, double ycen, double scale,
        uint32_t *dev_counts, uint32_t *colors){ 
    if (Version == 2) total_time += display_double_cuda(xcen, ycen, scale, dev_counts, colors);
    else total_time += display_double_single(xcen, ycen, scale, dev_counts, colors);
    instances++;
}

void swapVersion(void) {
    fflush(stdout);
#ifdef BENCHMARK
    double speedup;
    if (Version == 0) {
        single_core = total_time/instances; 
        printf("Avg time for Single: %.6f   Instances: %d\tSpeedup: 1\n", total_time/instances, instances);
    } else {
        speedup = single_core/(total_time/instances);
        if (Version == 1) printf("Avg time for OpenMP: %.6f   Instances: %d\tSpeedup: %.2f\n", total_time/instances, instances, speedup);
        else printf("Avg time for CUDA:   %.6f   Instances: %d\tSpeedup: %.2f\n", total_time/instances, instances, speedup);
    } 
#endif

    if (++Version > 2) Version = 0;

#ifdef SHOW_X
    XTextProperty tp;
    char name1[128] = "Mandelbrot Single Core";
    char name2[128] = "Mandelbrot OpenMP";
	char name3[128] = "Mandelbrot CUDA";
	char *n;
    if (Version == 0) n = name1;
    else if (Version == 1) n = name2;
    else n = name3;
	Status st = XStringListToTextProperty(&n, 1, &tp);
	if (st) XSetWMName(dpy, win, &tp);
#endif
}

void usage(){

    printf("Usage: benchmark [n] [m] [dim] [max_iter]\n");

    printf("\tn\t\t=\tnumber of blocks (defaults to 512)\n");

    printf("\tm\t\t=\tthreads per block (defaults to 512)\n");

    printf("\tdim\t\t=\twidth/height of canvas in pixels (defaults to 800)\n");

    printf("\tmax_iter\t=\tmax iterations (defaults to 100)\n\n");

    exit(1);
}

int main(int argc, char** argv){
    if(argc < 2){
        usage();
        return 0;
    }

    hipError_t err = hipSuccess;
    printf("%s", argv[0]);
    if (argc >= 2){
        n = atoi(argv[1]);
        printf(" %s", argv[1]);
    }
    if (argc >= 3){ 
        m = atoi(argv[2]);
        printf(" %s", argv[2]);
    }
    if (argc >= 4){
        dim = atoi(argv[3]);
        printf(" %s", argv[3]);
    }
    if (argc >= 5){
        max_iter = atoi(argv[4]);
        printf(" %s", argv[4]);
    }
    // if (argc >= 6){
    //     mem_mode = atoi(argv[5]);
    //     printf(" %s", argv[5]);
    // }
    size_t color_size = (max_iter +1) * sizeof(uint32_t);
    colors = (uint32_t *) malloc(color_size);
    hipMalloc((void**)&device_colors, color_size);
    double xcen = -0.5;
    double ycen = 0;
    double scale = 3;
    printf("\n");
    

#ifdef SHOW_X
	init_x11();
#endif
reset:
#ifdef BENCHMARK
    total_time = 0;
    instances = 0;
#endif
	init_colours();
    hipMemcpy(device_colors, colors, color_size, hipMemcpyHostToDevice);

    uint32_t *device_counts = NULL;
    size_t img_size = dim * dim * sizeof(uint32_t);
    err = hipMalloc(&device_counts, img_size);
    checkErr(err, (char*)"Failed to allocate dev_counts");
    uint32_t* host_counts = (uint32_t*)malloc(img_size);
    if (host_counts == 0) printf("Failed to allocate host_counts\n");

    uint32_t *dev_colors, *dev_counts;
    if (Version == 2) {
        dev_counts = device_counts;
        dev_colors = device_colors;
    } else {
        dev_counts = host_counts;
        dev_colors = colors;
    }
#ifdef BENCHMARK
#ifndef SHOW_X
    for (int i = 0; i < BENCHMARK_ITERATIONS; i++)
        display_double(xcen, ycen, scale, dev_counts, dev_colors);
    if (Version < 2) {
        swapVersion();
        goto reset;
    }
    swapVersion();
    return 0;
#endif
#endif

	display_double(xcen, ycen, scale, dev_counts, dev_colors);

#ifdef SHOW_X
    int getXMotion = 0;
    int getYMotion = 0;
	while(1) {
		XEvent event;
		KeySym key;
		char text[255];
		
		XNextEvent(dpy, &event);
        while (XPending(dpy) > 0)
            XNextEvent(dpy, &event);
		/* Just redraw everything on expose */
		if ((event.type == Expose) && !event.xexpose.count){
			XPutImage(dpy, win, gc, bitmap,
				0, 0, 0, 0,
				dim, dim);
		}

        // scroll to zoom
        if (event.type==ButtonPress) {
            if (event.xbutton.button == 4) {
                scale *= 0.8;
                display_double(xcen, ycen, scale, dev_counts, dev_colors);
            } else if (event.xbutton.button == 5) {
                scale *= 1.20;
                display_double(xcen, ycen, scale, dev_counts, dev_colors);
            }
        }

        // lock positions on drag
        if (event.type==ButtonPress) {
            if (event.xbutton.button == 3) {
                getXMotion = event.xbutton.x;
                getYMotion = event.xbutton.y;
            }
        }

        else if (event.type == MotionNotify) {
            xcen += (getXMotion-event.xbutton.x) * scale / dim;
            ycen += (getYMotion-event.xbutton.y) * scale / dim;
            display_double(xcen, ycen, scale, dev_counts, dev_colors);
            getXMotion = event.xbutton.x;
            getYMotion = event.xbutton.y;
        }
		
		/* Press 'x' to exit */
		if ((event.type == KeyPress) &&
			XLookupString(&event.xkey, text, 255, &key, 0) == 1)
			if (text[0] == 'x') break;

        /* Press 'k' to switch */
		if ((event.type == KeyPress) &&
			XLookupString(&event.xkey, text, 255, &key, 0) == 1)
			if (text[0] == 'k') {
                swapVersion();
                //exit_x11();
                goto reset;
            }

        /* Press 'r' to refresh */
		if ((event.type == KeyPress) &&
			XLookupString(&event.xkey, text, 255, &key, 0) == 1)
			if (text[0] == 'r') display_double(xcen, ycen, scale, dev_counts, dev_colors);

		/* Press 'a' to go left */
		if ((event.type == KeyPress) &&
			XLookupString(&event.xkey, text, 255, &key, 0) == 1)
			if (text[0] == 'a'){
                xcen -= 20 * scale / dim;
                display_double(xcen, ycen, scale, dev_counts, dev_colors);
            }

		/* Press 'w' to go up */
		if ((event.type == KeyPress) &&
			XLookupString(&event.xkey, text, 255, &key, 0) == 1)
			if (text[0] == 'w'){
                ycen -= 20 * scale / dim;
                display_double(xcen, ycen, scale, dev_counts, dev_colors);
            }

		/* Press 's' to go down */
		if ((event.type == KeyPress) &&
			XLookupString(&event.xkey, text, 255, &key, 0) == 1)
			if (text[0] == 's'){
                ycen += 20 * scale / dim;
                display_double(xcen, ycen, scale, dev_counts, dev_colors);
            }

		/* Press 'd' to go right */
		if ((event.type == KeyPress) &&
			XLookupString(&event.xkey, text, 255, &key, 0) == 1)
			if (text[0] == 'd'){
                xcen += 20 * scale / dim;
                display_double(xcen, ycen, scale, dev_counts, dev_colors);
            }

		/* Press 'q' to zoom out */
		if ((event.type == KeyPress) &&
			XLookupString(&event.xkey, text, 255, &key, 0) == 1)
			if (text[0] == 'q'){
                scale *= 1.25;
                display_double(xcen, ycen, scale, dev_counts, dev_colors);
            }

		/* Press 'e' to zoom in */
		if ((event.type == KeyPress) &&
			XLookupString(&event.xkey, text, 255, &key, 0) == 1)
			if (text[0] == 'e'){
                scale *= .80;
                display_double(xcen, ycen, scale, dev_counts, dev_colors);
            }

		/* Or simply close the window */
		if ((event.type == ClientMessage) &&
			((Atom) event.xclient.data.l[0] == wmDeleteMessage))
			break;
	}

    exit_x11();
#endif


    hipFree(dev_counts);
    hipFree(dev_colors);
    free(colors);
    free(host_counts);

	return 0;
}
